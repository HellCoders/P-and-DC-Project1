#include <iostream>
#include <math.h>
#include <ctime>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

struct stats {
    double mean;
    double min;
    double max;
    double stddev;
};

    
// CPU function to find mean of an array
double cpu_get_mean(int n, double *x) {
    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += x[i];
    }
    return sum/n;
}

// use CPU to calculate std deviation (Welford's algorithm)
double cpu_get_stddev(int n, double *x){
    double mean = x[0];
    double m2 = 0;
    double delta;
    double delta2;
    for (int i = 1; i < n; i++){
        delta = x[i] - mean;
        mean += delta/(i+1);
        delta2 = x[i] - mean;
        m2 += delta * delta2;
    }
    return sqrt(m2/n);
}

// CPU function to find max element of an array
double cpu_get_max(int n, double *x) {
    double max = x[0];
    for (int i = 1; i < n; i++) {
        max = (max < x[i]) ? x[i] : max;
    }
    return max;
}

// CPU function to find min element of an array
double cpu_get_min(int n, double *x) {
    double min = x[0];
    for (int i = 1; i < n; i++) {
        min = (x[i] < min) ? x[i] : min;
    }
    return min;
}

// use CPU to calculate min, mean, max, std deviation (Welford's algorithm)
stats cpu_get_all(int n, double *x){
    stats myStats; 
    double mean = x[0];
    double min = x[0];
    double max = x[0];
    double m2 = 0;
    double delta;
    double delta2;
    for (int i = 1; i < n; i++){
        max = (max < x[i]) ? x[i] : max;
        min = (x[i] < min) ? x[i] : min;
        delta = x[i] - mean;
        mean += delta/(i+1);
        delta2 = x[i] - mean;
        m2 += delta * delta2;
    }
    myStats.mean = mean;
    myStats.min = min;
    myStats.max = max;
    myStats.stddev = sqrt(m2/n);
    return myStats;
}

// Kernel function to find the maximum element of an array
__global__ void get_gpu_max(int n, double *x, double *results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double max = x[index];
    for (int i = index + stride; i < n; i += stride) {
        max = (max < x[i]) ? x[i] : max;
    }
    results[index] = max;
}

// Kernel function to find the minimum element of an array
__global__ void get_gpu_min(int n, double *x, double *results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double min = x[index];
    for (int i = index + stride; i < n; i += stride) {
        min = (x[i] < min) ? x[i] : min;
    }
    results[index] = min;
}

// kernel to calculate the mean on the GPU
__global__ void get_gpu_mean(int n, double *x, double *results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double mean = x[index];
    int count = 1;
    for (int i = index + stride; i < n; i += stride){
        count++;
        mean += (x[i] - mean)/count;
    }
    results[index] = mean;
}

// Calculate std deviation on the GPU
__global__ void get_gpu_stddev(int n, double *x, double *results){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double mean = x[index];
    double m2 = 0;
    double delta;
    double delta2;
    int count = 1;
    for (int i = index + stride; i < n; i += stride){
        count++;
        delta = x[i] - mean;
        mean += delta/count;
        delta2 = x[i] - mean;
        m2 += delta * delta2;
    }
    results[index] = m2;
}



// caluclate all stats on the GPU
__global__ void get_gpu_all(int n, double *x, stats *all_results){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double mean = x[index];
    double min = x[index];
    double max = x[index];
    double m2 = 0;
    double delta;
    double delta2;
    int count = 1;
    for (int i = index + stride; i < n; i += stride){
        max = (max < x[i]) ? x[i] : max;
        min = (x[i] < min) ? x[i] : min;
        count++;
        delta = x[i] - mean;
        mean += delta/count;
        delta2 = x[i] - mean;
        m2 += delta * delta2;
    }
    all_results[index].mean = mean;
    all_results[index].min = min;
    all_results[index].max = max;
    all_results[index].stddev = m2; // m2 not actually std dev
}

void print_diff(double x, double y){
    cout << "Difference: " << 100*(y - x)/x << "%\n";
}


void run_tests(int N_pre, int N_BLOCKS, int THREADS_PER_BLK) {

    // We need N to be a multiple of N_THREADS
    int N = N_BLOCKS * THREADS_PER_BLK * floor(N_pre / (THREADS_PER_BLK * N_BLOCKS));
    
    /**  
    cout << "N = " << N << endl;
    cout << "N_BLOCKS = " << N_BLOCKS << endl;
    cout << "THREADS_PER_BLK = " << THREADS_PER_BLK << endl;
    cout << "Allocating memory and initializing...";
    **/
    double *x;
    hipMallocManaged(&x, N*sizeof(double));
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
      x[i] = ((double) rand()) / ((double) RAND_MAX);
    }
    double *results;
    hipMallocManaged(&results, N_BLOCKS*THREADS_PER_BLK*sizeof(double));

    // use CPU to calculate max
    auto start = std::chrono::high_resolution_clock::now();
    double cpu_max = cpu_get_max(N, x);
    auto end = std::chrono::high_resolution_clock::now();
    auto dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "CPU calculated max:" << fixed << cpu_max << "_____";
    // fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,max,cpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",cpu_max);


    // use GPU to calculate max
    start = std::chrono::high_resolution_clock::now();
    get_gpu_max<<<N_BLOCKS, THREADS_PER_BLK>>>(N, x, results);
    hipDeviceSynchronize();
    double gpu_max = results[0];
    for (int i = 1; i < N_BLOCKS*THREADS_PER_BLK; i++) {
        gpu_max = (gpu_max < results[i]) ? results[i] : gpu_max;
    }
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "GPU calculated max:" << fixed << gpu_max << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,max,gpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",gpu_max);

    //print_diff(cpu_max, gpu_max);
    //cout << endl;

    // use CPU to calculate min
    start = std::chrono::high_resolution_clock::now();
    double cpu_min = cpu_get_min(N, x);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "CPU calculated min:" << fixed << cpu_min << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,min,cpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",cpu_min);
    // use GPU to calculate min
    start = std::chrono::high_resolution_clock::now();
    get_gpu_min<<<N_BLOCKS, THREADS_PER_BLK>>>(N, x, results);
    hipDeviceSynchronize();
    double gpu_min = results[0];
    for (int i = 1; i < N_BLOCKS*THREADS_PER_BLK; i++) {
        gpu_min = (results[i] < gpu_min) ? results[i] : gpu_min;
    }
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "GPU calculated min:" << fixed << gpu_min << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,min,gpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",gpu_min);
    //print_diff(cpu_min, gpu_min);
    //cout << endl;

    // use CPU to calculate mean
    start = std::chrono::high_resolution_clock::now();
    double cpu_mean = cpu_get_mean(N, x);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "CPU calculated mean:" << fixed << cpu_mean << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,avg,cpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",cpu_mean);
    // use GPU to calculate mean
    start = std::chrono::high_resolution_clock::now();
    get_gpu_mean<<<N_BLOCKS, THREADS_PER_BLK>>>(N, x, results);
    hipDeviceSynchronize();
    double gpu_mean_sum = 0;
    for (int i = 0; i < N_BLOCKS*THREADS_PER_BLK; i++) {
        gpu_mean_sum += results[i];
    }
    double gpu_mean = gpu_mean_sum/(N_BLOCKS*THREADS_PER_BLK);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "GPU calculated mean:" << fixed << gpu_mean << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,avg,gpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",gpu_mean);
    //print_diff(cpu_mean, gpu_mean);
    //cout << endl;

    // use CPU to calculate std dev
    start = std::chrono::high_resolution_clock::now();
    double cpu_stddev = cpu_get_stddev(N, x);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "CPU calculated std dev:" << fixed << cpu_stddev << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,dev,cpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    //fprintf(stdout," ,%ld\n",cpu_stddev);
    // use GPU to calculate std dev
    start = std::chrono::high_resolution_clock::now();
    get_gpu_stddev<<<N_BLOCKS, THREADS_PER_BLK>>>(N, x, results);
    hipDeviceSynchronize();
    double gpu_m2 = 0;
    for (int i = 0; i < N_BLOCKS*THREADS_PER_BLK; i++) {
        gpu_m2 += results[i];
    }
    double gpu_stddev = sqrt(gpu_m2/N);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    cout << "GPU calculated std dev:" << fixed << gpu_stddev << "_____";
    //fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    fprintf(stdout,"%d,%d,%d,dev,gpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    /**
    print_diff(cpu_stddev, gpu_stddev);
    cout << endl;
    **/
    //fprintf(stdout," ,%ld\n",gpu_stddev);
    // use CPU to calculate all stats
    start = std::chrono::high_resolution_clock::now();
    stats my_stats = cpu_get_all(N, x);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    /**
    cout << "Concurrent: CPU calculated max:" << fixed << my_stats.max << endl;
    cout << "Concurrent: CPU calculated min:" << fixed << my_stats.min << endl;
    cout << "Concurrent: CPU calculated mean:" << fixed << my_stats.mean << endl;
    cout << "Concurrent: CPU calculated std dev:" << fixed << my_stats.stddev << endl;
    fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    **/
    fprintf(stdout,"%d,%d,%d,all,cpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    

    hipFree(results);

    // use GPU to calculate all stats
    stats* all_results;
    hipMallocManaged(&all_results, N_BLOCKS*THREADS_PER_BLK*sizeof(stats));
    
    // start the timer
    start = std::chrono::high_resolution_clock::now();

    // run calculations on the GPU
    get_gpu_all<<<N_BLOCKS, THREADS_PER_BLK>>>(N, x, all_results);

    // synchrnonize 
    hipDeviceSynchronize();

    // We now need to accumulate results from all threads
    double m2 = all_results[0].stddev;
    double mean = all_results[0].mean;
    double delta;
    double new_mean;
    int n_a = N / (N_BLOCKS*THREADS_PER_BLK); 
    int n_b = n_a;
    double max = all_results[0].max;
    double min = all_results[0].min;
    for (int i = 1; i < N_BLOCKS*THREADS_PER_BLK; i++) {
        new_mean = all_results[i].mean;
        delta = new_mean - mean;

        // we update our running mean value
        mean = (n_a*mean + n_b*new_mean)/(n_a + n_b);

        m2 += all_results[i].stddev + delta * delta * n_a * n_b / (n_a + n_b);

        n_a += n_b;

        min = (all_results[i].min < min) ? all_results[i].min : min;
        max = (all_results[i].max > max) ? all_results[i].max : max;
    }
    double stddev = sqrt(m2/N);
    end = std::chrono::high_resolution_clock::now();
    dur_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start);
    /**
    cout << "Concurrent: GPU calculated max:" << fixed << max << endl;
    cout << "Concurrent: GPU calculated min:" << fixed << min << endl;
    cout << "Concurrent: GPU calculated mean:" << fixed << mean << endl;
    cout << "Concurrent: GPU calculated std dev:" << fixed << stddev << endl;
    fprintf(stdout, "Elapsed time %lld ns\n", dur_ns.count());
    **/
    fprintf(stdout,"%d,%d,%d,all,gpu,%lld\n",N,N_BLOCKS,THREADS_PER_BLK,dur_ns.count());
    
    // Free memory
    hipFree(x);
    hipFree(all_results);
}


int main(void) {

    // We want to display floats with max precision
    cout.precision(17);

    int Ns[] = {50000000,100000000,150000000};
    int TPBs[] = {256,512};
    int NBs[] = {1,4};

    for (int n : Ns) {
        for (int threads_per_block : TPBs) {
            for (int n_blocks : NBs) {
                run_tests(n, n_blocks, threads_per_block); 
            }
        }
    }
   
}
